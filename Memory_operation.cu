#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;

class Memory()
{
public:
    int N = 2<<20;
    int *a;
    size_t size = N*sizeof(int);

//for CPU
/* For CPU we will use 
1. malloc -> to create a memory into Heap.
2. free -> reference to pointer to free the space 
*/
void cpu()
{
//Using the left shift operator operation





//using a reference pointer to create a memory into Heap.
a = (int *) malloc(size);

free(a);
}

void gpu()
{
    int *a;
    hipMallocManaged(&a, size);
    hipFree(a);
}
};

int main()
{
    cpu();
    gpu();
    
    return 0;
}


/* 
iteration N

__global__ gpu()
{
    printf("", threadIdx.x);
}

int main()
{
    iteration N;
    gpu<<< 1, N>>>();
}