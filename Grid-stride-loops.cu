
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
CUDA provides a special variable giving the number of blocks 
in a grid, gridDim.x. Calculating the total number of 
threads in a grid then is simply the number of blocks in a grid multiplied by 
the number of threads in each block, gridDim.x * blockDim.x. 
*/

__global__ void kernel(int *a, int N)
{
    int indexWithinGrid = threadIdx.x + blockIdx.x * blockDim.x;

    int gridStride = gridDim.x * blockDim.x;

    for(int i = indexWithinGrid; i< N; i += gridStride)
    {
        //do work on a[i];
    }
}