#include "hip/hip_runtime.h"
#include<stdio.h>
#include< assert.h>
using namespace std;


//cuda error check
inline hipError_t checkCuda(hipError_t, result)

{
    if(result != hipSuccess)
    {
        printf(stdder, "CUDA Runtime Error: %s\n", cudaErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}



void initWith(float num, float *a, int N)
{
    for(int i= 0; i <N; ++i)
    {
        a[i] = num;
    }
}




__global__ void addVectorInto(float *result, float *a, float *b, int N)
{
    //Defining the size of the data index to avoid overflow.
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    //Defining the stride to devide the element operation equally of each threadIndex.
    int stride = blockDim.x * gridDim.x;

    for(int i=0; i<N; i+=stride)
    {
        result = a[i] + b[i];
    }
}

/* For cpu
We will convert the cpu operation to GPU program

void addVectorInto(float *result, float *a, float *b, int N)
{
    for(int i=0; i<N; ++i)
    {
        result[i] = a[i] + b[i];
    }
}


*/

//condition check
void checkElementsAre(float target, float *array, int N)
{
    for(int i=0; i<N; i++)
    {
        if(array[i] != target)
        {
            printf("Fail: array[%d] - %0.0f does not equal to %0.0f\n", i , array[i], target);
            exit(1);
        }
    }
    printf("Success!");
}


//main()

int main()
{
    const int N = 2<<20;  // bitwise shift operation
    size_t size = N*sizeof(float); // definging the size in HEAP.

    float *a;
    float *b;
    float *c;
/*
Changing the memory management function from CPU to GPU.
    //Defining the memory of each variable in Heap.
    a = (float *) malloc(size); // defining the partitions in Size(Heap)
    b = (float *) malloc(size);
    c = (float *) malloc(size);

    addVectorInto(c,a,b, N);
    checkElementsAre(7, c, N);

    free(a);
    free(b);
    free(c);
*/

checkCuda( hipMallocManaged( &a, size)));
checkCuda( hipMallocManaged( &b, size));
checkCuda( hipMallocManaged( &c, size));

size_t threadPerBlock;
size_t numberOfBlocks;
threadPerBlock = 256;

numberOfBlocks = (N + threadPerBlock -1) / threadPerBlock;

addVectorInto<<< numberOfBlocks, threadPerBlock>>>(c, a, b, N);

checkCuda( hipGetLastError() );
checkCuda( cudaDeviceSyncronize() );

checkElementsAre(7, c, N);

checkCuda( hipFree(a) );
checkCuda( hipFree(b) );
checkCuda( hipFree(c) );

}



}