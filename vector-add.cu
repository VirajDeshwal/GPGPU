#include "hip/hip_runtime.h"
#include<stdio.h>
#include<assert.h>

//Host function to initialize vector elements.

void initwith(float num, float *a, int N)
{
    for(int i=0; i<N; ++i)
    {
        a[i] = num;
    }
}

//kernel
__global__ void addVectorInto(float *result, float *a, float *b, int N)
{
    int index = threaIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    for(int i=0; i< N; i += stride)
    {
        result[i] = a[i]+b[i];
    }
}

//check elements
void checkElementsAre(float target, float *vector, int N)
{
    for(int i=0; i<N; i++)
    {
        if(vector[i] != target)
        {
            printf("Fail: vector[%d] - %0.0f does not equal to %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success.");
}

int main()

const int N = 2<<20;
size_t size = N *sizeof(float);

float *a;
float *b;
float *c;

hipMallocManaged( (&a, size) );
hipMallocManaged( (&b, size) );
hipMallocManaged( (&c, size) );

initWith(3, a, N);
initWith(4, b, N);
initWith(0, c, N);

size_t threadPerBlock = 1;
size_t numberOfBlocks = 1;

hipError_t addVectorsErr;
hipError_t asyncErr;

addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

addVectorsErr = hipGetLastError();
if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

asyncErr = hipDeviceSynchronize();
if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

checkElementsAre(7, c, N);

hipFree(a);
hipFree(b);
hipFree(c);
}

