#include<stdio.h>


//hipError_t type to define error variable

hipError_t err;
err = hipMallocManaged(&a, N);

if(err != hipSuccess)
{
    printf("Error: %s\n", hipGetErrorString(err));
}

/*
Launching kernels, which are defined to return void, 
do not return a value of type hipError_t. 
To check for errors occuring at the time of a kernel 
launch, for example if the launch configuration is erroneous, 
CUDA provides the hipGetLastError function, 
which does return a value of type hipError_t.
*/

/*
 * The macro can be wrapped around any function returning
 * a value of type `hipError_t`.
 */

 checkCuda( hipDeviceSynchronize())
}
